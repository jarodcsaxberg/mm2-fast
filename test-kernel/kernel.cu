#include "hip/hip_runtime.h"
#include <wb.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <fstream>
#include <immintrin.h>
#include <sys/mman.h>
#include <stdio.h>
#include <stdlib.h>
//#include "LISA-hash/lisa_hash.h"

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                                \
    if (err != hipSuccess) {                                              \
      wbLog(ERROR, "HIP error: ", hipGetErrorString(err));                \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define THREAD_SIZE 256

typedef uint64_t rmi_key_t;

enum query_state {
    GUESS_RMI_ROOT,
    GUESS_RMI_LEAF,
    LAST_MILE
};

typedef struct batchMetadata {
  int64_t qid;
  query_state state;
  rmi_key_t key;
  int64_t modelIndex;
  int64_t first;
  int64_t m;
} BatchMetadata;

//@@ Define constant memory for device kernel here
__constant__ rmi_key_t constant_sorted_array[65536 / sizeof(rmi_key_t)];

//@@ Helper functions for the rmi lookup
__device__ int64_t FCLAMP(double inp, double bound) {
  if (inp < 0.0) return 0;
  return (inp > bound ? bound : (size_t)inp);
}

__device__ int64_t get_guess_root_step(rmi_key_t key, double L0_PARAMETER0, double L0_PARAMETER1, int64_t L1_SIZE) {
  int64_t modelIndex;
  double fpred = std::fma(L0_PARAMETER1, key, L0_PARAMETER0);
  modelIndex = FCLAMP(fpred, L1_SIZE - 1.0);
  return modelIndex;
}

__device__ int64_t get_guess_leaf_step(rmi_key_t key, int64_t modelIndex, int64_t *err, int64_t n, double* L1_PARAMETERS) {
  double fpred = std::fma(L1_PARAMETERS[modelIndex * 3 + 1], key, L1_PARAMETERS[modelIndex * 3]);
  *err = *((uint64_t*) (L1_PARAMETERS + (modelIndex * 3 + 2)));
  int64_t guess = FCLAMP(fpred, n - 1.0);
  return guess;
}

__device__ void last_mile_search_one_step(rmi_key_t key, int64_t &first, int64_t &m) {
  int64_t half = m >> 1;
  int64_t middle = first + half;
  int64_t cond = (key >= constant_sorted_array[middle]);
  first = middle * cond + first * (1 - cond);
  m = (m - half) * cond + half * (1 - cond);
}

__device__ int process_query_one_step(
  BatchMetadata *bm,
  int64_t *pos, 
  int64_t n,
  double L0_PARAMETER0, 
  double L0_PARAMETER1, 
  int64_t L1_SIZE,
  double *L1_PARAMETERS
) {
  if(bm->state == GUESS_RMI_ROOT){
    bm->modelIndex = get_guess_root_step(bm->key, L0_PARAMETER0, L0_PARAMETER1, L1_SIZE);
    bm->state = GUESS_RMI_LEAF;
    // _mm_prefetch((const char *)(&L1_PARAMETERS[bm->modelIndex * 3]), _MM_HINT_T0);
    // _mm_prefetch((const char *)(&L1_PARAMETERS[bm->modelIndex * 3 + 2]), _MM_HINT_T0);
  } else if(bm->state == GUESS_RMI_LEAF) {
    int64_t err;
    int64_t guess = get_guess_leaf_step(bm->key, bm->modelIndex, &err, n, L1_PARAMETERS);
    bm->first = guess - err;
    if(bm->first < 0) bm->first = 0;
    int64_t last = guess + err + 1;
    if(last > n) last = n;
    bm->m = last - bm->first;
    bm->state = LAST_MILE;
    int64_t middle = bm->m >> 1;
    //_mm_prefetch((const char *)(&sorted_array[bm->first + middle]), _MM_HINT_T0);
  } else {
    if(bm->m > 1)
    {
      last_mile_search_one_step(bm->key, bm->first, bm->m);
      int64_t middle = bm->m >> 1;
      //_mm_prefetch((const char *)(&sorted_array[bm->first + middle]), _MM_HINT_T0);
    }
    if(bm->m == 1)
    {
      *pos = bm->first;

      if(constant_sorted_array[*pos] != bm->key)
        *pos = -1;
        
        return 0;
    }
  }
  return 1;
}

//@@ Insert kernel code here
__global__ void rmi_lookup(
  rmi_key_t *inputKeys, 
  double *inputL1_PARAMETERS,
  int64_t *outputPositions, 
  int64_t n,
  double L0_PARAMETER0, 
  double L0_PARAMETER1, 
  int64_t L1_SIZE
) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if(tid < n) {
    BatchMetadata bm;
    bm.qid = tid;
    bm.state = GUESS_RMI_ROOT;
    bm.key = inputKeys[tid];
    int64_t pos;
    int status = 1;

    //printf("tid %" PRId64 ", bm.state %d, bm.key %" PRIu64 "\n", bm.qid, bm.state, bm.key);
    do {
      status = process_query_one_step(&bm, &pos, n, L0_PARAMETER0, L0_PARAMETER1, L1_SIZE, inputL1_PARAMETERS);
    } while(status);

    outputPositions[tid] = pos;
    //printf("tid %d done, pos = %" PRId64 "\n", tid, pos);
  }
}

bool load_sorted_array(char* path, rmi_key_t **sorted_array, int64_t *n) {
  std::ifstream infile(path, std::ios::in | std::ios::binary);
  if (!infile.good()) {
    printf("%s file not found\n", path);
    exit(0);
  }

  infile.read((char *)(n), sizeof(uint64_t));
  *sorted_array = (rmi_key_t*) malloc((*n) * sizeof(rmi_key_t));
  if (*sorted_array == NULL) return false;
  
  infile.read((char*)(*sorted_array), (*n) * sizeof(rmi_key_t));
  if (!infile.good()) return false;
  
  return true;
}

bool load_rmi(char* path, double* L0_PARAMETER0, double* L0_PARAMETER1, int64_t* L1_SIZE, double** L1_PARAMETERS) {
  std::ifstream infile(path, std::ios::in | std::ios::binary);
  if (!infile.good()) {
    printf("%s file not found\n", path);
    exit(0);
  }

  infile.read((char *)(L0_PARAMETER0), sizeof(double));
  infile.read((char *)(L0_PARAMETER1), sizeof(double));
  infile.read((char *)(L1_SIZE), sizeof(int64_t));

  if (!infile.good()) {
    fprintf(stderr, "failed L0 params and L1_SIZE\n");
    return false;
  }

  *L1_PARAMETERS = (double*) malloc(*L1_SIZE * 3 * sizeof(double));
  if (*L1_PARAMETERS == NULL) {
    fprintf(stderr, "failed malloc\n");
    return false;
  };

  infile.read((char*)(*L1_PARAMETERS), *L1_SIZE * 3 * sizeof(double));
  // this fails in "normal" use case, assume that is okay
  if (!infile.good()) {
    fprintf(stderr, "failed read\n");
    //return false;
  }

  return true;
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int keysLength;
  rmi_key_t *hostKeys;
  int64_t *hostPositions;
  // host RMI values
  rmi_key_t *sorted_array;
  int64_t n;
  double L0_PARAMETER0 = 0.0;
  double L0_PARAMETER1 = 0.0;
  int64_t L1_SIZE = 0;
  double *L1_PARAMETERS;

  rmi_key_t *deviceKeys;
  double* deviceL1_PARAMETERS;
  int64_t *devicePositions;

  // hardcoded value for now
  char uint64_path[71] = "test/input/MT-human.fa_map-ont_minimizers_key_value_sorted_keys.uint64";

  // load_sorted_array with CPU
  if(!load_sorted_array(uint64_path, &sorted_array, &n)){
    fprintf(stderr, "Failed load_sorted_array\n");
    free(sorted_array);
    exit(-1);
  }
  fprintf(stderr, "Success load_sorted_array, n = %" PRId64 "\n", n);

  // hardcoded value for now
  char rmiparams_path[79] = "test/input/MT-human.fa_map-ont_minimizers_key_value_sorted_keys.rmi_PARAMETERS";

  // load_rmi with CPU
  if(!load_rmi(rmiparams_path, &L0_PARAMETER0, &L0_PARAMETER1, &L1_SIZE, &L1_PARAMETERS)) {
    fprintf(stderr, "Failed load_rmi\n");
    free(sorted_array);
    free(L1_PARAMETERS);
    exit(-2);
  }
  fprintf(stderr, "Success load_rmi\n");

  fprintf(stderr, "L0_PARAMETER0 = %E, L0_PARAMETER1 = %E, L1_SIZE = %ld\n", 
          L0_PARAMETER0, L0_PARAMETER1, L1_SIZE);

  args = wbArg_read(argc, argv);

  // wbImport does not work with uint64_t values, must load manually
  // this is certainly a bottleneck that must be improved upon
  fprintf(stderr, "loading keys...\n");
  std::ifstream infile("test/input/keys.raw", std::ios::in | std::ios::binary);
  std::string line;
  if (getline(infile, line)) { // get the first line (size) and malloc based on this value
    keysLength = std::stoi(line);
    hostKeys = (rmi_key_t*) malloc(keysLength * sizeof(rmi_key_t));
  }

  // Read the rest of the lines and populate the array
  int j = 0;
  while (getline(infile, line)) {
    hostKeys[j] = std::stoull(line);
    j++;
  }
  infile.close();

  hostPositions = (int64_t*) malloc(keysLength * sizeof(int64_t));

  //@@ Allocate GPU memory here
  wbCheck(hipMalloc(&deviceKeys, keysLength * sizeof(rmi_key_t)));
  wbCheck(hipMalloc(&deviceL1_PARAMETERS, L1_SIZE * 3 * sizeof(double)));
  wbCheck(hipMalloc(&devicePositions, keysLength * sizeof(int64_t)));

  //@@ Copy input and kernel to GPU here
  wbCheck(hipMemcpy(deviceKeys, hostKeys, keysLength * sizeof(rmi_key_t), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceL1_PARAMETERS, L1_PARAMETERS, L1_SIZE * 3 * sizeof(double), hipMemcpyHostToDevice));
  wbCheck(hipMemcpyToSymbol(HIP_SYMBOL(constant_sorted_array), sorted_array, n * sizeof(float)));

  //@@ Initialize grid and block dimensions here
  dim3 threads(THREAD_SIZE);
  dim3 blocks((keysLength + THREAD_SIZE - 1) / THREAD_SIZE);

  //@@ Launch the GPU kernel here
  uint64_t start_time = __rdtsc();
  fprintf(stderr, "TIMER LOG: rmi_lookup start- %" PRIu64 "\n", start_time);
  rmi_lookup<<<blocks, threads>>>(deviceKeys, deviceL1_PARAMETERS, devicePositions, n, L0_PARAMETER0, L0_PARAMETER1, L1_SIZE);
  hipDeviceSynchronize();
  uint64_t end_time = __rdtsc();
  uint64_t runtime = end_time - start_time;
  fprintf(stderr, "TIMER LOG: rmi_lookup end- %" PRIu64 "\n", end_time);
  fprintf(stderr, "TIMER LOG: rmi_lookup time- %" PRIu64 "\n", runtime);

  fprintf(stderr, "TIMER LOG: warmup start- %l" PRIu64 "\n", __rdtsc());
  rmi_lookup<<<blocks, threads>>>(deviceKeys, deviceL1_PARAMETERS, devicePositions, n, L0_PARAMETER0, L0_PARAMETER1, L1_SIZE);
  rmi_lookup<<<blocks, threads>>>(deviceKeys, deviceL1_PARAMETERS, devicePositions, n, L0_PARAMETER0, L0_PARAMETER1, L1_SIZE);
  hipDeviceSynchronize();
  fprintf(stderr, "TIMER LOG: warmup end- %l" PRIu64 "\n", __rdtsc());

  fprintf(stderr, "TIMER LOG: timing start- %l" PRIu64 "\n", __rdtsc());
  float totalTime = 0;
  float milliseconds = 0;
  hipEvent_t start, stop;
  wbCheck(hipEventCreate(&start));
  wbCheck(hipEventCreate(&stop));
  // Measure the average time of the kernel over 10 iterations
  wbCheck(hipEventRecord(start, 0));
  for (int i = 0; i < 10; ++i) {
    rmi_lookup<<<blocks, threads>>>(deviceKeys, deviceL1_PARAMETERS, devicePositions, n, L0_PARAMETER0, L0_PARAMETER1, L1_SIZE);
  }
  wbCheck(hipEventRecord(stop, 0));
  wbCheck(hipEventSynchronize(stop));
  wbCheck(hipEventElapsedTime(&milliseconds, start, stop));
  totalTime += milliseconds;
  fprintf(stderr, "Total time: %f\n", totalTime);
  fprintf(stderr, "TIMER LOG: timing end- %l" PRIu64 "\n", __rdtsc());

  //@@ Copy the device memory back to the host here
  wbCheck(hipMemcpy(hostPositions, devicePositions, keysLength * sizeof(int64_t), hipMemcpyDeviceToHost));

  // uncomment to get output values in slurm file
  // for(int i = 0; i < keysLength; i++) {
  //   fprintf(stderr, "positions[%d] = %" PRId64 "\n", i, hostPositions[i]);
  // }

  //@@ Solution
  wbSolution(args, hostPositions, keysLength);

  //@@ Free device memory
  wbCheck(hipFree(deviceKeys));
  wbCheck(hipFree(deviceL1_PARAMETERS));
  wbCheck(hipFree(devicePositions));

  //@@ Free host memory
  free(hostKeys);
  free(sorted_array);
  free(L1_PARAMETERS);
  free(hostPositions);

  return 0;
}